#include "hip/hip_runtime.h"
#include <cmath>
#include <mutex>

#include <ATen/Context.h>
#include <ATen/ScalarOps.h>
#include <ATen/Tensor.h>
#include <ATen/core/Generator.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/Optional.h>
#include <torch/library.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>

#include "kernel_forward.h"
#include "pytorch_utils.h"

#define DISPATCH_BLOCKSIZE(VALUE_HEAD_DIM, FN)        \
  {                                                   \
    if (VALUE_HEAD_DIM <= 64) {                       \
      constexpr bool kIs64x64 = true;                 \
      constexpr bool kSingleValueIteration = true;    \
      FN();                                           \
    } else {                                          \
      constexpr bool kIs64x64 = false;                \
      if (VALUE_HEAD_DIM <= 128) {                    \
        constexpr bool kSingleValueIteration = true;  \
        FN();                                         \
      } else {                                        \
        constexpr bool kSingleValueIteration = false; \
        FN();                                         \
      }                                               \
    }                                                 \
  }

#define DISPATCH_KERNEL(QUERY, KEY, VALUE, FUNC)                              \
  {                                                                           \
    hipDeviceProp_t* properties =                                              \
        at::cuda::getDeviceProperties(QUERY.device().index());                \
    const int computeCapability = properties->major * 10 + properties->minor; \
    DISPATCH_BLOCKSIZE(                                                       \
        VALUE.size(-1), ([&]() {                                              \
          static constexpr int64_t kQueriesPerBlock = kIs64x64 ? 64 : 32;     \
          static constexpr int64_t kKeysPerBlock = kIs64x64 ? 64 : 128;       \
          DISPATCH_TYPES(                                                     \
              QUERY, ([&]() {                                                 \
                DISPATCH_ARCHTAG(                                             \
                    computeCapability, ([&]() {                               \
                      using AlignedAK = AttentionKernel<                      \
                          scalar_t,                                           \
                          ArchTag,                                            \
                          true,                                               \
                          kQueriesPerBlock,                                   \
                          kKeysPerBlock,                                      \
                          kSingleValueIteration>;                             \
                      /* Run a more efficient kernel (with `isAligned=True`)  \
                      if memory is correctly aligned*/                        \
                      bool isAligned =                                        \
                          (QUERY.stride(2) % AlignedAK::kAlignmentQ == 0 &&   \
                           KEY.stride(2) % AlignedAK::kAlignmentK == 0 &&     \
                           VALUE.stride(2) % AlignedAK::kAlignmentV == 0);    \
                      /* TODO: Should we warn or log somewhere when we use a  \
                      less efficient kernel due to wrong alignment? */        \
                      DISPATCH_BOOL(isAligned, kIsAligned, ([&]() {           \
                                      using Kernel = AttentionKernel<         \
                                          scalar_t,                           \
                                          ArchTag,                            \
                                          kIsAligned,                         \
                                          kQueriesPerBlock,                   \
                                          kKeysPerBlock,                      \
                                          kSingleValueIteration>;             \
                                      FUNC();                                 \
                                    }))                                       \
                    }))                                                       \
              }));                                                            \
        }));                                                                  \
  }

namespace {
template <typename scalar_t>
struct TypeTraits;

template <>
struct TypeTraits<cutlass::half_t> {
  using scalar_t = cutlass::half_t;

  static constexpr __host__ at::ScalarType atScalarType() {
    return at::ScalarType::Half;
  }
  template <int nDim>
  static __host__ at::PackedTensorAccessor32<scalar_t, nDim> packed_accessor(
      at::Tensor const& tensor) {
    return at::PackedTensorAccessor32<scalar_t, nDim>(
        (scalar_t*)(tensor.data_ptr()),
        tensor.sizes().data(),
        tensor.strides().data());
  }
};

template <>
struct TypeTraits<cutlass::bfloat16_t> {
  using scalar_t = cutlass::bfloat16_t;

  static constexpr __host__ at::ScalarType atScalarType() {
    return at::ScalarType::BFloat16;
  }
  template <int nDim>
  static __host__ at::PackedTensorAccessor32<scalar_t, nDim> packed_accessor(
      at::Tensor const& tensor) {
    return at::PackedTensorAccessor32<scalar_t, nDim>(
        (scalar_t*)(tensor.data_ptr()),
        tensor.sizes().data(),
        tensor.strides().data());
  }
};

template <>
struct TypeTraits<float> {
  using scalar_t = float;

  static constexpr __host__ at::ScalarType atScalarType() {
    return at::ScalarType::Float;
  }
  template <int nDim>
  static __host__ at::PackedTensorAccessor32<scalar_t, nDim> packed_accessor(
      at::Tensor const& tensor) {
    return tensor.packed_accessor32<scalar_t, nDim>();
  }
};

/*
  There are 2 modes for using this function.
  (Mode BMHK) With all the heads having the same seqlen
  (Mode 1MHK) `batch=1` with all tokens across batches concatenated
*/
std::tuple<at::Tensor, at::Tensor, int64_t, int64_t>
efficient_attention_forward_cutlass(
    const at::Tensor& query, // [b, seqlen, num_heads, K]
    const at::Tensor& key, // [b, seqlen, num_heads, K]
    const at::Tensor& value, // [b, seqlen, num_heads, Kv]
    const c10::optional<at::Tensor>& bias, // [b, num_heads, seqlen, seqlen]
    // (Mode 1MHK only) [b+1]: cu_seqlens_q[b] contains the
    // position of the first query token for batch $b
    const c10::optional<at::Tensor>& cu_seqlens_q,
    // (Mode 1MHK only) [b+1]: cu_seqlens_k[b] contains the
    // position of the first key token for batch $b
    const c10::optional<at::Tensor>& cu_seqlens_k,
    // (Mode 1MHK only) Maximum sequence length across batches
    const c10::optional<int64_t> max_seqlen_q_,
    double dropout_p, // attention matrix dropout probability
    bool compute_logsumexp,
    bool causal,
    c10::optional<double> scale) {
#ifdef XFORMERS_MEM_EFF_ATTENTION_DISABLE_FORWARD
  TORCH_CHECK(
      false,
      "MemoryEfficient build has been disabled at build time with -DXFORMERS_MEM_EFF_ATTENTION_DISABLE_FORWARD");
#else
  at::globalContext().alertNotDeterministic(
      "efficient_attention_forward_cutlass");

  TORCH_CHECK(query.dim() == 4);
  TORCH_CHECK(key.dim() == 4);
  TORCH_CHECK(value.dim() == 4);

  // Batch sizes
  TORCH_CHECK(query.size(0) == key.size(0));
  TORCH_CHECK(query.size(0) == value.size(0));

  // Sequence length
  TORCH_CHECK(key.size(1) == value.size(1));

  // Num heads
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(2) == value.size(2));

  // Embedding per head
  TORCH_CHECK(query.size(3) == key.size(3));

  int64_t max_seqlen_q, max_seqlen_k;
  TORCH_CHECK(cu_seqlens_q.has_value() == cu_seqlens_k.has_value());
  if (cu_seqlens_q.has_value()) {
    TORCH_CHECK(cu_seqlens_q->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_k->scalar_type() == at::ScalarType::Int);
    TORCH_CHECK(cu_seqlens_q->dim() == 1 && cu_seqlens_k->dim() == 1);
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_q));
    CHECK_NOSPARSE_CONTIGUOUS_CUDA((*cu_seqlens_k));
    TORCH_CHECK(cu_seqlens_q->size(0) == cu_seqlens_k->size(0));
    TORCH_CHECK(query.size(0) == 1, "cu_seqlen only supports batch_size=1");
    TORCH_CHECK(max_seqlen_q_.has_value());
    max_seqlen_q = *max_seqlen_q_;
    max_seqlen_k = 0; // Will be set inside the kernel
  } else {
    max_seqlen_q = query.size(1);
    max_seqlen_k = key.size(1);
  }

  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(query);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(key);
  CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA(value);

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t num_heads = query.size(-2);
  int64_t K = query.size(-1);
  int64_t Kv = value.size(-1);

  at::Tensor res;
  at::Tensor logsumexp;

  const bool use_dropout = std::fpclassify(dropout_p) != FP_ZERO;
  at::PhiloxCudaState rng_engine_inputs;
  if (use_dropout) {
    at::CUDAGeneratorImpl* gen =
        at::get_generator_or_default<at::CUDAGeneratorImpl>(
            c10::nullopt, at::cuda::detail::getDefaultCUDAGenerator());

    std::lock_guard<std::mutex> lock(gen->mutex_);
    // if using dropout, we produce 1 random number for each element of the
    // attention tensor
    rng_engine_inputs = gen->philox_cuda_state(B * num_heads * M * N);
  }

  auto launchKernel = [&](auto _k, int computeCapability) {
    using Kernel = decltype(_k);
    using scalar_t = typename Kernel::scalar_t;
    (void)_k;

    res = at::empty(
        {B, M, num_heads, Kv},
        query.options().dtype(
            TypeTraits<typename Kernel::output_t>::atScalarType()));

    // NOTE: Should be aligned (by padding) in case M is
    // not a good number for loading during backward
    constexpr decltype(M) kAlignLSE = Kernel::kAlignLSE;
    logsumexp = at::empty(
        {cu_seqlens_q.has_value() ? cu_seqlens_q->size(0) - 1 : B,
         num_heads,
         compute_logsumexp ? ceil_div(max_seqlen_q, kAlignLSE) * kAlignLSE : 0},
        query.options().dtype(at::ScalarType::Float));

    typename Kernel::Params p;
    p.query_ptr = (scalar_t*)query.data_ptr();
    p.key_ptr = (scalar_t*)key.data_ptr();
    p.value_ptr = (scalar_t*)value.data_ptr();
    p.logsumexp_ptr = compute_logsumexp
        ? (typename Kernel::lse_scalar_t*)logsumexp.data_ptr()
        : nullptr;
    at::Tensor output_accum;
    if (Kernel::kNeedsOutputAccumulatorBuffer) {
      output_accum = at::empty(
          {B, M, num_heads, Kv},
          query.options().dtype(
              TypeTraits<typename Kernel::output_accum_t>::atScalarType()));
      p.output_accum_ptr =
          (typename Kernel::output_accum_t*)output_accum.data_ptr();
    } else {
      p.output_accum_ptr = nullptr;
    }
    p.output_ptr = (typename Kernel::output_t*)res.data_ptr();

    if (cu_seqlens_q.has_value()) {
      p.cu_seqlens_q_ptr = (int32_t*)cu_seqlens_q->data_ptr();
      p.cu_seqlens_k_ptr = (int32_t*)cu_seqlens_k->data_ptr();
    }

    p.num_heads = num_heads;
    p.head_dim = query.size(3);
    p.head_dim_value = value.size(3);
    p.num_queries = max_seqlen_q;
    p.num_keys = max_seqlen_k;
    p.num_batches = cu_seqlens_q.has_value() ? cu_seqlens_q->size(0) - 1 : B;
    p.causal = causal;
    if (scale.has_value()) {
      p.scale = float(*scale);
    } else {
      p.scale = float(1.0 / std::sqrt(float(p.head_dim)));
    }

    ASSIGN_CHECK_OVERFLOW(p.q_strideB, query.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.k_strideB, key.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.v_strideB, value.stride(0));
    ASSIGN_CHECK_OVERFLOW(p.q_strideM, query.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.k_strideM, key.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.v_strideM, value.stride(1));
    ASSIGN_CHECK_OVERFLOW(p.q_strideH, query.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.k_strideH, key.stride(2));
    ASSIGN_CHECK_OVERFLOW(p.v_strideH, value.stride(2));

    if (bias.has_value()) {
      CHECK_NOSPARSE_LASTCONTIGUOUS_CUDA((*bias));
      p.attn_bias_ptr = (scalar_t*)bias->data_ptr();

      // assign strides for bias, viewed as
      // (batch_sz, n_heads, n_queries, n_keys)
      const at::Tensor bias_4d_view =
          get_bias_4d_view(*bias, B, num_heads, M, N);
      ASSIGN_CHECK_OVERFLOW(p.bias_strideB, bias_4d_view.stride(0));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideH, bias_4d_view.stride(1));
      ASSIGN_CHECK_OVERFLOW(p.bias_strideM, bias_4d_view.stride(2));
    }

    p.use_dropout = use_dropout;
    if (p.use_dropout) {
      p.rng_engine_inputs = rng_engine_inputs;
      p.dropout_prob = dropout_p;
    }

    constexpr auto kernel_fn = attention_kernel_batched<Kernel>;
    size_t smem_bytes = sizeof(typename Kernel::SharedStorage);
    if (smem_bytes > 0xc000) {
      TORCH_INTERNAL_ASSERT(
          computeCapability >= 70,
          "This kernel requires too much shared memory on this machine!");
      AT_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
          kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes));
    }
    Kernel::check_supported(p);
    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream>>>(p);
  };
  // Dispatch to the right kernel
  DISPATCH_KERNEL(query, key, value, ([&]() {
                    launchKernel(Kernel{}, computeCapability);
                  }));

  AT_CUDA_CHECK(hipGetLastError());

  // uint64_t -> int64_t bitwise casting as PyTorch don't support uint64_t
  // so just fake it as a int64_t
  int64_t seed, offset;
  if (use_dropout) {
    std::memcpy(&seed, &rng_engine_inputs.seed_, sizeof(seed));
    std::memcpy(&offset, &rng_engine_inputs.offset_.val, sizeof(offset));
  }

  return std::make_tuple(res, logsumexp, seed, offset);
#endif
}
} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention_forward_cutlass"),
      TORCH_FN(efficient_attention_forward_cutlass));
}
